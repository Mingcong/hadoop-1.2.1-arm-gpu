#include "hip/hip_runtime.h"
/***********************************************************************
 	hadoop-gpu
	Authors: Koichi Shirahata, Hitoshi Sato, Satoshi Matsuoka

This software is licensed under Apache License, Version 2.0 (the  "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
-------------------------------------------------------------------------
File: gpu-matmul.cc
 - Plain matrix multiplication on GPU.
Version: 0.20.1
***********************************************************************/

#include  "stdint.h"

#include "hadoop/Pipes.hh"
#include "hadoop/TemplateFactory.hh"
#include "hadoop/StringUtils.hh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>

#include <time.h>
#include <sys/time.h>

__global__ void mul(float *a, float *b, float *muled, int len)
{
  int i;
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int nthreads = blockDim.x * gridDim.x;
  int part = len / nthreads;

  for(i = part*tid; i < part*(tid+1); i++) {
    muled[i] = a[i] * b[i];
  }
  return;
}

class MatmulMap: public HadoopPipes::Mapper {
public:
  MatmulMap(HadoopPipes::TaskContext& context){}

  double gettime() {
    struct timeval tv;
    gettimeofday(&tv,NULL);
    return tv.tv_sec+tv.tv_usec * 1e-6;
  }

  void map(HadoopPipes::MapContext& context) {
    double st, ft, t, ifi, kf, mf, hdf, dhf;
    st = gettime();
    int k;
    std::string line = context.getInputValue();
    std::vector<std::string> elements = HadoopUtils::splitString(line, " ");
    int i = HadoopUtils::toFloat(elements[0]);
    int j = HadoopUtils::toFloat(elements[1]);
    int T = (elements.size()-2) / 2;
    float a[T], b[T], vals[T];

    //variables for CUDA
    float *ad, *bd, *muled;
    size_t array_size = sizeof(float) * T;

    t = gettime();

    std::string key = HadoopUtils::toString(i) + " " + HadoopUtils::toString(j);
    
    for(k =  0; k < T; ++k) {
      a[k] = HadoopUtils::toFloat(elements[k + 2]);
    }
    for(k = 0; k < T; ++k) {
      b[k] = HadoopUtils::toFloat(elements[k + (T + 2)]);
    }    
    ifi = gettime();

    hipMalloc((void **)&ad, array_size);
    hipMalloc((void **)&bd, array_size);
    hipMalloc((void **)&muled, array_size);    
    
    mf = gettime();

    hipMemcpy(ad, a, array_size, hipMemcpyHostToDevice);
    hipMemcpy(bd, b, array_size, hipMemcpyHostToDevice);

    hdf = gettime();

    mul<<<16, 512>>>(ad, bd, muled, T);

    kf = gettime();
    
    hipMemcpy(vals, muled, array_size, hipMemcpyDeviceToHost);

    dhf = gettime();

    for(k = 0; k < T; ++k) {
      context.emit(key, HadoopUtils::toString(vals[k]));
    }
    ft = gettime();
    std::cout << ft-st << ", " << t-st << ", " << ifi-t << ", " << mf-ifi << ", " << dhf-mf << 
	", " << kf-hdf << ", " << dhf-kf << ", " << ft-dhf << std::endl;

    std::cout << key << std::endl;
  }
};

class MatmulReduce: public HadoopPipes::Reducer {
public:
  MatmulReduce(HadoopPipes::TaskContext& context){}
  void reduce(HadoopPipes::ReduceContext& context) {
    // sumup values which have the same keys
    float sum = 0;
    while (context.nextValue()) {
      sum += HadoopUtils::toFloat(context.getInputValue());
    }
    context.emit(context.getInputKey(), HadoopUtils::toString(sum));
  }
};

int main(int argc, char *argv[]) {
  return HadoopPipes::runTask(HadoopPipes::TemplateFactory<MatmulMap,
                                                           MatmulReduce>());
}
