#include "hip/hip_runtime.h"
/***********************************************************************
 	hadoop-gpu
	Authors: Koichi Shirahata, Hitoshi Sato, Satoshi Matsuoka

This software is licensed under Apache License, Version 2.0 (the  "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
-------------------------------------------------------------------------
File: gpu-matmul.cc
 - Plain matrix multiplication on GPU.
Version: 0.20.1
***********************************************************************/

#include  "stdint.h"

#include "hadoop/Pipes.hh"
#include "hadoop/TemplateFactory.hh"
#include "hadoop/StringUtils.hh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>

#include <time.h>
#include <sys/time.h>

/*
__global__ void mul(float *a, float *b, float *muled, int len)
{
  int i;
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int nthreads = blockDim.x * gridDim.x;
  int part = len / nthreads;
  for(i = part*tid; i < part*(tid+1); i++) {
    muled[i] = a[i] * b[i];
  }
  return;
}
*/

__global__ void mul(float *a, float *b, float *muled)
{
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  muled[i] = a[i] * b[i];
  return;
}

class MatmulMap: public HadoopPipes::Mapper {
public:
  MatmulMap(HadoopPipes::TaskContext& context){}


  void map(HadoopPipes::MapContext& context) {
    int k;
    std::string line = context.getInputValue();
    std::vector<std::string> elements = HadoopUtils::splitString(line, " ");
    int i = HadoopUtils::toFloat(elements[0]);
    int T = (elements.size()-1) / 2;
    float a[T], b[T], vals[T];

    //variables for CUDA
    float *ad, *bd, *muled;
    size_t array_size = sizeof(float) * T;


    std::string key = HadoopUtils::toString(i);
    
    for(k =  0; k < T; ++k) {
      a[k] = HadoopUtils::toFloat(elements[k + 1]);
    }
    for(k = 0; k < T; ++k) {
      b[k] = HadoopUtils::toFloat(elements[k + (T + 1)]);
    }    

    hipMalloc((void **)&ad, array_size);
    hipMalloc((void **)&bd, array_size);
    hipMalloc((void **)&muled, array_size);    
    

    hipMemcpy(ad, a, array_size, hipMemcpyHostToDevice);
    hipMemcpy(bd, b, array_size, hipMemcpyHostToDevice);


    mul<<<T/512, 512>>>(ad, bd, muled);

    
    hipMemcpy(vals, muled, array_size, hipMemcpyDeviceToHost);


    for(k = 0; k < T; ++k) {
      context.emit(key, HadoopUtils::toString(vals[k]));
    }
  }
};

class MatmulReduce: public HadoopPipes::Reducer {
public:
  MatmulReduce(HadoopPipes::TaskContext& context){}
  void reduce(HadoopPipes::ReduceContext& context) {
    // sumup values which have the same keys
    float sum = 0;
    while (context.nextValue()) {
      sum += HadoopUtils::toFloat(context.getInputValue());
    }
    context.emit(context.getInputKey(), HadoopUtils::toString(sum));
  }
};

int main(int argc, char *argv[]) {
  return HadoopPipes::runTask(HadoopPipes::TemplateFactory<MatmulMap,
                                                           MatmulReduce>());
}
